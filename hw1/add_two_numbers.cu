
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int a, b, c;
    a = 2;
    b= 7; 
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);
    
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    add<<<1, 1>>>(d_a, d_b, d_c);
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    printf("a + b = %d\n", c);    
}