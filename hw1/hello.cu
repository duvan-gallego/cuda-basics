
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
    printf("Hello from GPU");
    #if __CUDA_ARCH__ >= 200
        printf("Hi Cuda World");
    #endif
}

int main() {
    printf("beforeDeviceFunc\n\n");
    hello<<<1, 1>>>();
    hipError_t cudaerr = hipDeviceSynchronize();
    printf(hipGetErrorString(cudaerr));
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
    printf("after DeviceFunc\n\n"); 
    return 0; 
}
