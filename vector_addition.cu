
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void vadd(int *a, int *b, int *c) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;    
    c[index] = a[index] + b[index];
}

int main() {
    int *a, *b, *c; 
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);
    
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    a = (int *)malloc(size);    
    b = (int *)malloc(size);     
    c = (int *)malloc(size);

    for (int i = 0; i < N; i++) {
        a[i] = rand()/N;
        b[i] = rand()/N;
    }

    // Copy input data from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch kernel
    vadd<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

    // Copy output data from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Check result
    for (int i = 0; i < 10; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Cleanup
    free(a); free(b); free(c); 
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    return 0;
}